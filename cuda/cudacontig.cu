#include "hip/hip_runtime.h"
// To compile - gcc -o 3dFDTD FDTD3D.c -lm

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define IMAX 100;
#define JMAX 100;
#define KMAX 100;


// This was taken from stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define CHECK_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        printf("GPU error: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

/**
 *  Second half of the total loop circuit.
 */
extern __global__ void loop2_GPU(
        double (*Ez)[IMAX][JMAX], 
        double (*Hx)[IMAX][JMAX], 
        double (*Hy)[IMAX][JMAX], 
        double (*Hz)[IMAX][JMAX],
        double Da, double Db) {
    int i, j;
    int k = blockIdx.x * 32 + threadIdx.x;

    if (k < KMAX && k > 0) {
        for (j = 0; j < JMAX-1; j++) {
            for (i = 1; i < IMAX-1; i++) {
                Hx[i][j][k] = Da*Hx[i][j][k] + Db*((Ez[i][j][k] - Ez[i][j+1][k]) + (Ez[i][j][k+1]-Ez[i][j][k]));
            }
        }
       for (j = 1; j < JMAX-1; j++) {
           for (i = 0; i < IMAX-1; i++) {
               Hy[i][j][k] = Da*Hy[i][j][k] + Db*((Ez[i+1][j][k] - Ez[i][j][k]) + (Ez[i][j][k]-Ez[i][j][k+1]));
           }
       }
    }
    if (k < KMAX) {
       for (j = 0; j < JMAX-1; j++) {
           for (i = 0; i < IMAX-1; i++) {
               Hz[i][j][k] = Da*Hz[i][j][k] + Db*((Ez[i][j][k] - Ez[i+1][j][k]) + (Ez[i][j+1][k]-Ez[i][j][k]));
           }
       }
    }
}

int main() {
    int nmax = 1000, nhalf = 20, no = nhalf*3;
    int i, j, n,k;
    double c = 2.99792458e8, pi = 3.141592654, sigma = 0, mu = 4.0 * pi * 1.0e-7, eps = 8.85418782e-12;
    double delta = 1e-3;
    double dt = delta/(c*1.41421356237);

    double *Ex, *Ey, *Ez, *Hy, *Hx, *Hz;

    hipEvent_t start_event, stop_event;
    float elapsed_time;

    Ex = (double *)calloc((imax+1) * (jmax+1) * (kmax+1), sizeof(double));
    Ey = (double *)calloc((imax+1) * (jmax+1) * (kmax+1), sizeof(double));
    Ez = (double *)calloc((imax+1) * (jmax+1) * (kmax+1), sizeof(double));
    Hx = (double *)calloc((imax+1) * (jmax+1) * (kmax+1), sizeof(double));
    Hy = (double *)calloc((imax+1) * (jmax+1) * (kmax+1), sizeof(double));
    Hz = (double *)calloc((imax+1) * (jmax+1) * (kmax+1), sizeof(double));

    double* g_Ex;
    double* g_Ey;
    double* g_Ez;
    double* g_Hx;
    double* g_Hy;
    double* g_Hz;
    CHECK_ERROR(hipMalloc((void**)&g_Ex, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double)));
    CHECK_ERROR(hipMalloc((void**)&g_Ey, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double)));
    CHECK_ERROR(hipMalloc((void**)&g_Ez, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double)));
    CHECK_ERROR(hipMalloc((void**)&g_Hx, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double)));
    CHECK_ERROR(hipMalloc((void**)&g_Hy, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double)));
    CHECK_ERROR(hipMalloc((void**)&g_Hz, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double)));

    double Ca,Cb,Da,Db;

    Ca = (1-((sigma*dt)/(2*eps)))/(1+((sigma*dt)/(2*eps)));
    Cb = (dt/(eps*delta))/(1+((sigma*dt)/(2*eps)));
    Da = (1-((sigma*dt)/(2*mu)))/(1+((sigma*dt)/(2*mu)));
    Db = (dt/(mu*delta))/(1+((sigma*dt)/(2*mu)));

    CHECK_ERROR(hipEventCreate(&start_event));
    CHECK_ERROR(hipEventCreate(&stop_event));
    CHECK_ERROR(hipEventRecord(start_event, 0));

    CHECK_ERROR(hipMemcpy(g_Ex, Ex, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(g_Ey, Ey, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(g_Ez, Ez, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(g_Hx, Hx, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(g_Hy, Hy, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(g_Hz, Hz, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyHostToDevice));

    for (n = 0; n < nmax; n++) {
        //for (k = 1; k < kmax; k++) {
        //    for (j = 1; j < jmax; j++) {
        //        for (i = 0; i < imax; i++) {
        //            Ex[i][j][k] = Ca*Ex[i][j][k] + Cb*((Hz[i][j][k] - Hy[i][j-1][k]) + (Hy[i][j][k-1] - Hy[i][j][k]));
        //        }
        //    }
        //}
        //for (k = 1; k < kmax; k++) {
        //    for (j = 0; j < jmax; j++) {
        //        for (i = 1; i < imax; i++) {
        //            Ey[i][j][k] = Ca*Ey[i][j][k] + Cb*((Hz[i-1][j][k] - Hy[i][j][k]) + (Hy[i][j][k] - Hy[i][j][k-1]));
        //        }
        //    }
        //}
        //for (k = 0; k < kmax; k++) {
        //    for (j = 1; j < jmax; j++) {
        //        for (i = 1; i < imax; i++) {
        //            Ez[i][j][k] = Ca*Ez[i][j][k] + Cb*((Hz[i][j][k] - Hy[i-1][j][k]) + (Hy[i][j-1][k] - Hy[i][j][k]));
        //        }
        //    }
        //}
        //Ez[imax/2][jmax/2][kmax/2] = exp(-(pow(((n-no)/(double)nhalf),2.0)));

        dim3 threadsPerBlock(32);
        dim3 numBlocks((kmax + threadsPerBlock.x-1) / threadsPerBlock.x);
        loop2_GPU<<<numBlocks, threadsPerBlock>>>(g_Ez, g_Hx, g_Hy, g_Hz, Da, Db);
    }

    CHECK_ERROR(hipMemcpy(Ex, g_Ex, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_ERROR(hipMemcpy(Ey, g_Ey, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_ERROR(hipMemcpy(Ez, g_Ez, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_ERROR(hipMemcpy(Hx, g_Hx, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_ERROR(hipMemcpy(Hy, g_Hy, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_ERROR(hipMemcpy(Hz, g_Hz, (imax+1) * (jmax+1) * (kmax+1) * sizeof(double), hipMemcpyDeviceToHost));

    CHECK_ERROR(hipFree(g_Hx));
    CHECK_ERROR(hipFree(g_Hy));
    CHECK_ERROR(hipFree(g_Hz));
    CHECK_ERROR(hipFree(g_Ez));

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    printf("GPU Time: %.2f\n", elapsed_time);

    return 0;
}
